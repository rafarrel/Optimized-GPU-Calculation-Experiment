#include "hip/hip_runtime.h"
/**
 * Cuda Programming HW Problem One: Dot Product
 * Alex Farrell
*/
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 256
#define NUM_BLOCKS 256
#define N 65536


__global__
void dotp(float *u, float *v, float *partialSum, int n) {
    // Partial sums
    __shared__ float localCache[BLOCK_SIZE];
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    localCache[threadIdx.x] = u[tidx] * v[tidx];
    __syncthreads();

    // Parallel reduction
    int cacheIndex = threadIdx.x;
    int i = blockDim.x / 2;
    while (i > 0) {
        if (cacheIndex < i) {
            localCache[cacheIndex] = localCache[cacheIndex] + localCache[cacheIndex + i];
            // Need to be careful calling __syncthreads in an if
            // statements to avoid issues such as deadlock. Should
            // be safe for this problem.
            __syncthreads();
        }
        i = i / 2;
    }

    if (cacheIndex == 0) {
        partialSum[blockIdx.x] = localCache[cacheIndex];
    }
}

//----------------------------------------------------------------

int main() {
    srand48(time(0));

    float *U, *V, *partialSum;
    float *dev_U, *dev_V, *dev_partialSum;

    U = (float *) malloc(N * sizeof(float));
    V = (float *) malloc(N * sizeof(float));
    partialSum = (float *) malloc(N * sizeof(float));

    hipMalloc((void **) &dev_U, N*sizeof(float));
    hipMalloc((void **) &dev_V, N*sizeof(float));
    hipMalloc((void **) &dev_partialSum, N*sizeof(float));

    //----------------------------------------------------------------
    // GPU Calculation

    // Testing vectors
    for (int i=0; i<N; ++i) {
        U[i] = (float) drand48();
        V[i] = (float) drand48();
    }

    // *** NOTE: The CUDA events are for getting the elapsed time for the GPU calculation. ***
    hipEvent_t startWithMemGPU, stopWithMemGPU;
    hipEvent_t startNoMemGPU_1, stopNoMemGPU_1;
    hipEvent_t startNoMemGPU_2, stopNoMemGPU_2;
    hipEventCreate(&startWithMemGPU);
    hipEventCreate(&stopWithMemGPU);
    hipEventRecord(startWithMemGPU, 0);

    // Copies data to the GPU so it can perform the calculation.
    hipMemcpy(dev_U, U, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_V, V, N*sizeof(float), hipMemcpyHostToDevice);

    hipEventCreate(&startNoMemGPU_1);
    hipEventCreate(&stopNoMemGPU_1);
    hipEventRecord(startNoMemGPU_1, 0);

    // GPU Calculation kernel
    dotp<<<NUM_BLOCKS, BLOCK_SIZE>>>(dev_U, dev_V, dev_partialSum, N);

    hipEventRecord(stopNoMemGPU_1, 0);
    hipEventSynchronize(stopNoMemGPU_1);

    hipDeviceSynchronize();
    hipMemcpy(partialSum, dev_partialSum, NUM_BLOCKS*sizeof(float), hipMemcpyDeviceToHost);

    hipEventCreate(&startNoMemGPU_2);
    hipEventCreate(&stopNoMemGPU_2);
    hipEventRecord(startNoMemGPU_2, 0);

    // Finish GPU calculation CPU-side by addding the partial sums.
    float gpuResult = 0.0;
    for (int i=0; i<NUM_BLOCKS; ++i) {
        gpuResult = gpuResult + partialSum[i];
    }

    hipEventRecord(stopNoMemGPU_2, 0);
    hipEventSynchronize(stopNoMemGPU_2);

    hipEventRecord(stopWithMemGPU, 0);
    hipEventSynchronize(stopWithMemGPU);

    // Elapsed Time
    float gpuElapsedWithMem;
    float gpuElapsedNoMem_1, gpuElapsedNoMem_2;
    float gpuElapsedNoMem_total = 0.0;
    hipEventElapsedTime(&gpuElapsedWithMem, startWithMemGPU, stopWithMemGPU);
    hipEventElapsedTime(&gpuElapsedNoMem_1, startNoMemGPU_1, stopNoMemGPU_1);
    hipEventElapsedTime(&gpuElapsedNoMem_2, startNoMemGPU_2, stopNoMemGPU_2);
    gpuElapsedNoMem_total = gpuElapsedNoMem_1 + gpuElapsedNoMem_2;

    hipEventDestroy(startWithMemGPU);
    hipEventDestroy(stopWithMemGPU);
    hipEventDestroy(startNoMemGPU_1);
    hipEventDestroy(stopNoMemGPU_1);
    hipEventDestroy(startNoMemGPU_2);
    hipEventDestroy(stopNoMemGPU_2);
    
    // CUDA Error Check
    hipError_t err = hipGetLastError();
    const char *msg = hipGetErrorName(err);
    printf("error = |%s|\n", msg);

    //----------------------------------------------------------------
    // CPU Calculation (for comparison to GPU Calculation)
    hipEvent_t startCPU, stopCPU;
    hipEventCreate(&startCPU);
    hipEventCreate(&stopCPU);
    hipEventRecord(startCPU, 0);

    float cpuResult = 0.0;
    for (int i=0; i<N; ++i) {
        cpuResult = cpuResult + (U[i] * V[i]);    
    }

    hipEventRecord(stopCPU, 0);
    hipEventSynchronize(stopCPU);

    float cpuElapsed;
    hipEventElapsedTime(&cpuElapsed, startCPU, stopCPU);
    hipEventDestroy(startCPU);
    hipEventDestroy(stopCPU);

    //----------------------------------------------------------------
    // Relative Error
    float relativeError = 0.0;
    relativeError = fabs((gpuResult - cpuResult) / gpuResult);

    //----------------------------------------------------------------
    // Results
    printf("GPU Result: %0.4f\n", gpuResult);
    printf("CPU Result: %0.4f\n", cpuResult);
    printf("Relative Error: %0.8f\n", relativeError);
    printf("CPU Elapsed Time: %0.4f\n", cpuElapsed);
    printf("GPU Elapsed Time (with mem copies): %0.4f\n", gpuElapsedWithMem);
    printf("GPU Elapsed Time (no mem copies): %0.4f\n", gpuElapsedNoMem_total);

    //----------------------------------------------------------------
    // Cleanup
    hipFree(dev_U);
    hipFree(dev_V);
    hipFree(dev_partialSum);

    free(U);
    free(V);
    free(partialSum);

    return 0;
}